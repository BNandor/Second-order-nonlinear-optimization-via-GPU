#include <iostream>
#include "core/AD/DDouble.cuh"

int main() {
    DFloat *dev_c;
    DFloat *c = (DFloat *) malloc(sizeof(DFloat));
    unsigned *dev_global_id;
    hipMalloc((void **) &dev_c, sizeof(DFloat));
    hipMalloc((void **) &dev_global_id, sizeof(unsigned));

    unsigned global_id_val = 0;
    hipMemcpy(dev_global_id, &global_id_val, sizeof(unsigned), hipMemcpyHostToDevice);
    test<<<1, 1>>>(dev_c, dev_global_id);

    hipMemcpy(c, dev_c, sizeof(DFloat), hipMemcpyDeviceToHost);
    hipFree(dev_c);
    hipFree(dev_global_id);
    free(c);
    return 0;
}
