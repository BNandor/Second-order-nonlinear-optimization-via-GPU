#include <iostream>
#include <iomanip>
#include <algorithm>

#include "core/common/Constants.cuh"
#include "core/optimizer/refine/LBFGS.cuh"
#include "core/optimizer/refine/GradientDescent.cuh"
#include "core/common/Random.cuh"
#include "core/common/Metrics.cuh"
#include "core/optimizer/perturb/DE/DEContext.h"
#include "core/common/OptimizerContext.cuh"
#include "core/common/model/BoundedParameter.cuh"
#include "core/optimizer/markov/OperatorMarkovChain.cuh"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <random>
#include <fstream>

void testOptimizer() {

    DEContext deContext = DEContext();
    GAContext gaContext = GAContext();

    OptimizerContext optimizerContext = OptimizerContext(deContext,gaContext);
    // Set f
    optimizerContext.model = SNLPModel(deContext);

    Metrics metrics = Metrics(optimizerContext.model,&optimizerContext.cudaMemoryModel.cudaConfig); //idempotent
    optimizerContext.getCurrentLocalSearch()->setupGlobalData(optimizerContext.getModelPopulationSize()); //idempotent
    optimizerContext.cudaMemoryModel.allocateFor(optimizerContext.model); // idempotent
    optimizerContext.cudaMemoryModel.copyModelToDevice(optimizerContext.model); // idempotent

    optimizerContext.model.loadModel(optimizerContext.cudaMemoryModel.dev_x, optimizerContext.cudaMemoryModel.dev_data,
                                     metrics); // idempotent
    metrics.getCudaEventMetrics().recordStartCompute(); // idempotent
    optimizerContext.cudaMemoryModel.cudaRandom.initialize(optimizerContext.getThreadsInGrid(), optimizerContext.getBlocksPerGrid(),
                          optimizerContext.getThreadsPerBlock()); // idempotent

    // EXECUTE KERNEL
    optimizerContext.cudaMemoryModel.initLoopPointers();

    optimizerContext.getCurrentPerturbator()->evaluateF(optimizerContext.cudaMemoryModel.cudaConfig,optimizerContext.cudaMemoryModel.dev_Model,
                                                        optimizerContext.cudaMemoryModel.dev_x1,
                                                        optimizerContext.cudaMemoryModel.dev_data,
                                                        optimizerContext.cudaMemoryModel.dev_F1);
    metrics.modelPerformanceMetrics.fEvaluations=1;
    metrics.modelPerformanceMetrics.markovIterations=0;
    OperatorMarkovChain markovChain=OperatorMarkovChain(&optimizerContext,&metrics);
    while(metrics.modelPerformanceMetrics.fEvaluations < optimizerContext.totalFunctionEvaluations) {
        markovChain.operate();
        markovChain.hopToNext();
    }

    metrics.getCudaEventMetrics().recordStopCompute();
    optimizerContext.cudaMemoryModel.copyModelsFromDevice(metrics.modelPerformanceMetrics);
    metrics.modelPerformanceMetrics.printBestModel(&optimizerContext.model);
    metrics.modelPerformanceMetrics.persistBestModelTo(&optimizerContext.model,std::string("finalModel") + std::string(OPTIMIZER::name) + std::string(".csv"));
    metrics.printFinalMetrics();
}

int main(int argc, char** argv) {
    testOptimizer();
    return 0;
}

// TODO make x1,x2,F1,F2 consistent in every operator (i.e evaluate F2 after every perturbation) DONE

// TODO add Simulated Annealing to hyper level
// TODO as a first step, skip mutating the operator parameters

