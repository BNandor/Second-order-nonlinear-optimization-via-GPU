#include "hip/hip_runtime.h"
#include <iostream>
#include "core/common/Tests.cuh"

void testDFloat() {
    DDouble *dev_c;
    DDouble *c = (DDouble *) malloc(sizeof(DDouble));
    unsigned *dev_global_id;
    hipMalloc((void **) &dev_global_id, sizeof(unsigned));
    hipMalloc((void **) &dev_c, sizeof(DDouble));


    unsigned global_id_val = 0;
    hipMemcpy(dev_global_id, &global_id_val, sizeof(unsigned), hipMemcpyHostToDevice);
    testDFloatKernel<<<1, 1>>>(dev_c, dev_global_id);

    hipMemcpy(c, dev_c, sizeof(DDouble), hipMemcpyDeviceToHost);
    assert(c->value == 36);
    hipFree(dev_c);
    hipFree(dev_global_id);
    free(c);
}

void testDFuncBFS() {


    unsigned *dev_global_id;
    hipMalloc((void **) &dev_global_id, sizeof(unsigned));
    unsigned global_id_val = 0;
    hipMemcpy(dev_global_id, &global_id_val, sizeof(unsigned), hipMemcpyHostToDevice);
    functionTestsKernel<<<1, 1>>>(dev_global_id);
    hipFree(dev_global_id);

}

void testF1() {
    unsigned *dev_global_id;
    hipMalloc((void **) &dev_global_id, sizeof(unsigned));
    unsigned global_id_val = 0;
    hipMemcpy(dev_global_id, &global_id_val, sizeof(unsigned), hipMemcpyHostToDevice);
    testF1DFloat<<<1, 1>>>(dev_global_id);
    hipFree(dev_global_id);
}

int main() {
//    testDFloat();
//    testDFuncBFS();
    testF1();
    return 0;
}

// Create the Function concept: ([DDouble a])-> compute parameter index order once (BFS), and propagate derivatives that way
// will have: orderArray[operatorTreeSize] - container indices of parameter in order
//            parameters[maxIndex]-contains references of DDouble parameters
// calculate local stack size limit,
// keep a min heap of size operatorTreeSize and a statistical vector to check for duplicates.
// OPT: keep the orderArray in shared memory, to reduce