#include <iostream>
#include <iomanip>
#include <algorithm>
//#define SAFE
//#define PRINT

//#define PROBLEM_ROSENBROCK2D
//#define PROBLEM_PLANEFITTING
//#define PROBLEM_SNLP
//#define PROBLEM_SNLP3D

//#define GLOBAL_SHARED_MEM

#include "core/common/Constants.cuh"
#include "core/optimizer/refine/LBFGS.cuh"
#include "core/optimizer/refine/GradientDescent.cuh"
#include "core/common/Random.cuh"
#include "core/common/Metrics.cuh"
#include "core/optimizer/perturb/DE/DEContext.h"
#include "core/common/OptimizerContext.cuh"
#include "core/common/model/BoundedParameter.cuh"
//#include "core/optimizer/perturb/GA/GeneticAlgorithm.cu"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <random>
#include <fstream>

void testOptimizer() {
    Random cudaRandom = Random();
    DEContext deContext = DEContext();
    GAContext gaContext = GAContext();

    OptimizerContext optimizerContext = OptimizerContext(deContext,gaContext);
    // Set f
    optimizerContext.model = SNLPModel(deContext);

    Metrics metrics = Metrics(optimizerContext.model); //idempotent
    optimizerContext.getCurrentLocalSearch()->setupGlobalData(optimizerContext.getModelPopulationSize()); //idempotent
    optimizerContext.cudaMemoryModel.allocateFor(optimizerContext.model); // idempotent
    optimizerContext.cudaMemoryModel.copyModelToDevice(optimizerContext.model); // idempotent

    optimizerContext.model.loadModel(optimizerContext.cudaMemoryModel.dev_x, optimizerContext.cudaMemoryModel.dev_data,
                                     metrics); // idempotent
    metrics.getCudaEventMetrics().recordStartCompute(); // idempotent
    cudaRandom.initialize(optimizerContext.getThreadsInGrid(), optimizerContext.getBlocksPerGrid(),
                          optimizerContext.getThreadsPerBlock()); // idempotent

    // EXECUTE KERNEL
    optimizerContext.cudaMemoryModel.initLoopPointers();
    optimizerContext.getCurrentPerturbator()->evaluateF(optimizerContext.cudaConfig,optimizerContext.cudaMemoryModel.dev_Model,
                                                        optimizerContext.cudaMemoryModel.dev_x1,
                                                        optimizerContext.cudaMemoryModel.dev_data,
                                                        optimizerContext.cudaMemoryModel.dev_F1);
    unsigned currentFEvaluations=1;
    unsigned currentGeneration=0;
    while(currentFEvaluations < optimizerContext.totalFunctionEvaluations) {
        //dev_F1 contains the costs of the current model
        //dev_x1 is the current model
        optimizerContext.getCurrentPerturbator()->perturb(optimizerContext.cudaConfig,
                                                          &optimizerContext.model,
                                                          optimizerContext.cudaMemoryModel.dev_Model,
                                                          optimizerContext.cudaMemoryModel.dev_x1,
                                                          optimizerContext.cudaMemoryModel.dev_x2,
                                                          optimizerContext.cudaMemoryModel.dev_data,
                                                          optimizerContext.cudaMemoryModel.dev_F1,
                                                          optimizerContext.cudaMemoryModel.dev_F2,
                                                          &cudaRandom);

        //dev_F2 contains the costs of the differential model
        //dev_x2 is the differential model
        optimizerContext.getCurrentLocalSearch()->optimize(optimizerContext.cudaMemoryModel.dev_x2, optimizerContext.cudaMemoryModel.dev_data, optimizerContext.cudaMemoryModel.dev_F2, optimizerContext.getCurrentLocalSearch()->getDevGlobalContext(),optimizerContext.cudaMemoryModel.dev_Model,optimizerContext.cudaConfig);
        //evaluated differential model into F2
        //select the best models from current and differential models
        optimizerContext.getCurrentSelector()->select(optimizerContext.cudaConfig,
                                                      optimizerContext.cudaMemoryModel.dev_x1,
                                                      optimizerContext.cudaMemoryModel.dev_x2,
                                                      optimizerContext.cudaMemoryModel.dev_F1,
                                                      optimizerContext.cudaMemoryModel.dev_F2);
        optimizerContext.getCurrentSelector()->printPopulationCostAtGeneration(optimizerContext.cudaConfig,optimizerContext.cudaMemoryModel.dev_F2,currentGeneration);

        optimizerContext.cudaMemoryModel.swapModels();
        std::for_each(optimizerContext.getCurrentOperators().begin(),optimizerContext.getCurrentOperators().end(),[&currentFEvaluations](auto op){
                currentFEvaluations+=op->fEvaluationCount();
        });

        ++currentGeneration;
        // dev_x1 contains the next models, dev_F1 contains the associated costs
    }

    metrics.getCudaEventMetrics().recordStopCompute();
    optimizerContext.cudaMemoryModel.copyModelsFromDevice(metrics.modelPerformanceMetrics);
    metrics.modelPerformanceMetrics.printBestModel(&optimizerContext.model);
    metrics.modelPerformanceMetrics.persistBestModelTo(&optimizerContext.model,std::string("finalModel") + std::string(OPTIMIZER::name) + std::string(".csv"));
    printf("\ntime ms : %f\n", metrics.getCudaEventMetrics().getElapsedKernelMilliSec());
}

int main(int argc, char** argv) {
    testOptimizer();
    return 0;
}

// TODO make x1,x2,F1,F2 consistent in every operator (i.e evaluate F2 after every perturbation) DONE

// TODO add Simulated Annealing to hyper level
// TODO as a first step, skip mutating the operator parameters

