#include "hip/hip_runtime.h"
#include <iostream>
#include "core/common/Tests.cuh"

void testDFloat() {
    DDouble *dev_c;
    DDouble *c = (DDouble *) malloc(sizeof(DDouble));
    unsigned *dev_global_id;
    hipMalloc((void **) &dev_global_id, sizeof(unsigned));
    hipMalloc((void **) &dev_c, sizeof(DDouble));


    unsigned global_id_val = 0;
    hipMemcpy(dev_global_id, &global_id_val, sizeof(unsigned), hipMemcpyHostToDevice);
    testDFloatKernel<<<1, 1>>>(dev_c, dev_global_id);

    hipMemcpy(c, dev_c, sizeof(DDouble), hipMemcpyDeviceToHost);
    assert(c->value == 36);
    hipFree(dev_c);
    hipFree(dev_global_id);
    free(c);
}

void testDFuncBFS() {


    unsigned *dev_global_id;
    hipMalloc((void **) &dev_global_id, sizeof(unsigned));
    unsigned global_id_val = 0;
    hipMemcpy(dev_global_id, &global_id_val, sizeof(unsigned), hipMemcpyHostToDevice);
    functionTestsKernel<<<1, 1>>>(dev_global_id);
    hipFree(dev_global_id);

}

void testF1() {
    unsigned xSize = 2;
    double *dev_x;
    hipMalloc((void **) &dev_x, xSize * sizeof(double));
    double x[2] = {100.0, 2.0};
    hipMemcpy(dev_x, &x, xSize * sizeof(double), hipMemcpyHostToDevice);
    testF1DFloat<<<1, 1>>>(dev_x, xSize);
    hipFree(dev_x);
}

void testPlaneFitting() {
    const unsigned xSize = 3;
    const unsigned dataSize = 6;
    double *dev_x;
    double *dev_dx;
    double *dev_F;
    double *dev_data;
    hipMalloc((void **) &dev_x, xSize * sizeof(double));
    hipMalloc((void **) &dev_dx, xSize * sizeof(double));
    hipMalloc((void **) &dev_F, sizeof(double));
    hipMalloc((void **) &dev_data, dataSize * sizeof(double));
    double x[xSize] = {5.5, 99.0, -1.0};
    double data[dataSize] = {1.0, 1.0, 1.0, 2.0, 0.0, 0.0};
    hipMemcpy(dev_x, &x, xSize * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_data, &data, dataSize * sizeof(double), hipMemcpyHostToDevice);
    testPlaneFitting<<<1, 2>>>(dev_x, dev_dx, dev_F, dev_data);
    hipFree(dev_x);
    hipFree(dev_dx);
    hipFree(dev_F);
    hipFree(dev_data);
}

int main() {
//    testDFloat();
//    testDFuncBFS();
//    testF1();
    testPlaneFitting();
    return 0;
}

// Create the Function concept: ([DDouble a])-> compute parameter index order once (BFS), and propagate derivatives that way
// will have: orderArray[operatorTreeSize] - container indices of parameter in order
//            parameters[maxIndex]-contains references of DDouble parameters
// calculate local stack size limit,
// keep a min heap of size operatorTreeSize and a statistical vector to check for duplicates.
// OPT: keep the orderArray in shared memory, to reduce