#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
//#define SAFE
//#define PRINT

//#define PROBLEM_ROSENBROCK2D
//#define PROBLEM_PLANEFITTING
//#define PROBLEM_SNLP
//#define PROBLEM_SNLP3D

//#define GLOBAL_SHARED_MEM

#include "core/common/Constants.cuh"
#include "core/optimizer/LBFGS.cuh"
#include "core/optimizer/GradientDescent.cuh"
#include "core/optimizer/DifferentialEvolution.cuh"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <random>
#include <fstream>

//void testDFloat() {
//    DDouble *dev_c;
//    DDouble *c = (DDouble *) malloc(sizeof(DDouble));
//    unsigned *dev_global_id;
//    hipMalloc((void **) &dev_global_id, sizeof(unsigned));
//    hipMalloc((void **) &dev_c, sizeof(DDouble));
//
//
//    unsigned global_id_val = 0;
//    hipMemcpy(dev_global_id, &global_id_val, sizeof(unsigned), hipMemcpyHostToDevice);
//    testDFloatKernel<<<1, 1>>>(dev_c, dev_global_id);
//
//    hipMemcpy(c, dev_c, sizeof(DDouble), hipMemcpyDeviceToHost);
//    assert(c->value == 36);
//    hipFree(dev_c);
//    hipFree(dev_global_id);
//    free(c);
//}
//
//void testDFuncBFS() {
//
//
//    unsigned *dev_global_id;
//    hipMalloc((void **) &dev_global_id, sizeof(unsigned));
//    unsigned global_id_val = 0;
//    hipMemcpy(dev_global_id, &global_id_val, sizeof(unsigned), hipMemcpyHostToDevice);
//    functionTestsKernel<<<1, 1>>>(dev_global_id);
//    hipFree(dev_global_id);
//
//}
//
//void testF1() {
//    unsigned xSize = 2;
//    double *dev_x;
//    hipMalloc((void **) &dev_x, xSize * sizeof(double));
//    double x[2] = {100.0, 2.0};
//    hipMemcpy(dev_x, &x, xSize * sizeof(double), hipMemcpyHostToDevice);
//    testF1DFloat<<<1, 1>>>(dev_x, xSize);
//    hipFree(dev_x);
//}
//void testQueue() {
//    unsigned xSize = 2;
//    double *dev_x;
//    hipMalloc((void **) &dev_x, xSize * sizeof(double));
//    double x[2] = {100.0, 2.0};
//    hipMemcpy(dev_x, &x, xSize * sizeof(double), hipMemcpyHostToDevice);
//    testQueue<<<1, 1>>>(dev_x);
//    hipFree(dev_x);
//}
//void testDot() {
//    unsigned xSize = 2;
//    double *dev_x;
//    hipMalloc((void **) &dev_x, xSize * sizeof(double));
//    double x[2] = {100.0, 2.0};
//    hipMemcpy(dev_x, &x, xSize * sizeof(double), hipMemcpyHostToDevice);
//    testDot<<<1, THREADS_PER_BLOCK>>>(dev_x);
//    hipFree(dev_x);
//}

void generateInitialPopulation(double *x, unsigned xSize) {
    std::uniform_real_distribution<double> unif(-10000, 10000);
    std::default_random_engine re(time(NULL));
    for (int i = 0; i < xSize; i++) {
        x[i] = unif(re);
    }
}
void readPopulation(double *x, unsigned xSize, std::string filename) {
    std::fstream input;
    input.open(filename.c_str());
    if (input.is_open()) {
        unsigned cData = 0;
        while (input >> x[cData]) {
            cData++;
        }
        std::cout << "read: " << cData << " expected: " << xSize
                  << std::endl;
        assert(cData == xSize);
    } else {
        std::cerr << "err: could not open " << filename << std::endl;
        exit(1);
    }
}

void generatePlanePoints(double A, double B, double C, double *data, unsigned pointCount) {
    std::uniform_real_distribution<double> unif(0, 1);
    std::default_random_engine re;
    std::normal_distribution<double> normal(0.0, 1);

    for (int i = 0; i < pointCount; i++) {
        data[i * RESIDUAL_CONSTANTS_DIM_1] = unif(re);
        data[i * RESIDUAL_CONSTANTS_DIM_1 + 1] = unif(re);
        data[i * RESIDUAL_CONSTANTS_DIM_1 + 2] =
                A * data[i * RESIDUAL_CONSTANTS_DIM_1] + B * data[i * RESIDUAL_CONSTANTS_DIM_1 + 1] + C + normal(re);
    }
}

#if defined(PROBLEM_SNLP) || defined(PROBLEM_SNLP3D)

void readSNLPProblem(double *data, std::string filename) {
    std::fstream input;
    input.open(filename.c_str());
    if (input.is_open()) {
        unsigned cData = 0;
        while (input >> data[cData]) {
            cData++;
        }
        std::cout << "read: " << cData << " expected: " << RESIDUAL_CONSTANTS_COUNT_1 * RESIDUAL_CONSTANTS_DIM_1
                  << std::endl;
        assert(cData == RESIDUAL_CONSTANTS_COUNT_1 * RESIDUAL_CONSTANTS_DIM_1);
    } else {
        std::cerr << "err: could not open " << filename << std::endl;
        exit(1);
    }
}

void readSNLPAnchors(double *data, std::string filename) {
    std::fstream input;
    input.open(filename.c_str());
    if (input.is_open()) {
        unsigned cData = 0;
        while (input >> data[cData]) {
            cData++;
        }
        std::cout << "read: " << cData << " expected: " << RESIDUAL_CONSTANTS_COUNT_2 * RESIDUAL_CONSTANTS_DIM_2
                  << std::endl;
        assert(cData == RESIDUAL_CONSTANTS_COUNT_2 * RESIDUAL_CONSTANTS_DIM_2);
    } else {
        std::cerr << "err: could not open " << filename << std::endl;
        exit(1);
    }
}

#endif
void persistBestSNLPModel(double *x, int modelSize, std::string filename) {
    std::ofstream output;
    output.open(filename.c_str());
    if (output.is_open()) {
        for (int i=0;i<modelSize;i++){
            output<<std::setprecision(17)<<x[i]<<std::endl;
        }
        output.close();
    } else {
        std::cout << "err: could not open " << filename << std::endl;
        exit(1);
    }
}

void testPlaneFitting() {

    hiprandState *dev_curandState;
    hipEvent_t start, stop, startCopy, stopCopy;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&startCopy);
    hipEventCreate(&stopCopy);


    const unsigned xSize = X_DIM * POPULATION_SIZE;

#if defined(PROBLEM_SNLP) || defined(PROBLEM_SNLP3D)
    const unsigned dataSize = RESIDUAL_CONSTANTS_DIM_1 * RESIDUAL_CONSTANTS_COUNT_1 +
                              RESIDUAL_CONSTANTS_DIM_2 * RESIDUAL_CONSTANTS_COUNT_2;
#else
    const unsigned dataSize = RESIDUAL_CONSTANTS_DIM_1 * RESIDUAL_CONSTANTS_COUNT_1;
#endif
//#ifdef GLOBAL_SHARED_MEM
    OPTIMIZER::GlobalData *dev_globalContext;// TODO POPULATION_SIZE number of shared contexts must be created (indexed by blockIndex)
    hipMalloc(&dev_globalContext, sizeof(OPTIMIZER::GlobalData)*POPULATION_SIZE/* TODO here have POPULATION_SIZE of these*/);
    printf("Allocating %lu global memory\n",sizeof(OPTIMIZER::GlobalData)*POPULATION_SIZE);
//#endif

    double *dev_x;
    double *dev_xDE;
    double *dev_x1;
    double *dev_x2;
    double *dev_data;
    double *dev_F;
    double *dev_FDE;
    double *dev_F1;
    double *dev_F2;



    // ALLOCATE DEVICE MEMORY
    hipMalloc((void **) &dev_x, xSize * sizeof(double));
    hipMalloc((void **) &dev_xDE, xSize * sizeof(double));
    hipMalloc((void **) &dev_data, dataSize * sizeof(double));
    hipMalloc((void **) &dev_F, POPULATION_SIZE * sizeof(double));
    hipMalloc((void **) &dev_FDE, POPULATION_SIZE * sizeof(double));
    hipMalloc(&dev_curandState, THREADS_PER_GRID * sizeof(hiprandState));

    // GENERATE PROBLEM
    double x[xSize] = {};
    double solution[xSize] = {};
    double finalFs[POPULATION_SIZE] = {};
    double data[dataSize] = {};

#ifdef PROBLEM_PLANEFITTING
    double A = -5.5;
    double B = 99;
    double C = -1;
    generatePlanePoints(A, B, C, data, RESIDUAL_CONSTANTS_COUNT_1);
    generateInitialPopulation(x, xSize);
#endif

#ifdef PROBLEM_ROSENBROCK2D
    data[0] = 1.0;
    data[1] = 100.0;
    x[0] = 100.0;
    x[1] = 2.0;
//    generateInitialPopulation(x, xSize);
#endif

#if defined(PROBLEM_SNLP) || defined(PROBLEM_SNLP3D)
    readSNLPProblem(data, PROBLEM_PATH);

    readSNLPAnchors(data + RESIDUAL_CONSTANTS_DIM_1 * RESIDUAL_CONSTANTS_COUNT_1,
                    PROBLEM_ANCHOR_PATH);
//    generateInitialPopulation(x, xSize);
    readPopulation(x, xSize,PROBLEM_INPUT_POPULATION_PATH);
#endif
    // COPY TO DEVICE
    hipEventRecord(startCopy);
    hipMemcpy(dev_x, &x, xSize * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_data, &data, dataSize * sizeof(double), hipMemcpyHostToDevice);
    hipEventRecord(stopCopy);
    hipEventRecord(start);

    // EXECUTE KERNEL
    // initialize hiprand
    setupCurand<<<POPULATION_SIZE, THREADS_PER_BLOCK>>>(dev_curandState);
    dev_x1 = dev_x;
    dev_x2 = dev_xDE;
    dev_F1 = dev_F;
    dev_F2 = dev_FDE;

#if  defined(OPTIMIZER_MIN_INIT_DE) || defined(OPTIMIZER_MIN_DE)
    OPTIMIZER::optimize<<<POPULATION_SIZE, THREADS_PER_BLOCK>>>(dev_x1, dev_data,dev_F1, dev_globalContext);
#endif

#ifdef OPTIMIZER_SIMPLE_DE
    OPTIMIZER::evaluateF<<<POPULATION_SIZE, THREADS_PER_BLOCK>>>(dev_x1, dev_data, dev_F1, dev_globalContext);
#endif

    for (unsigned i = 0; i < DE_ITERATION_COUNT; i++) {
        differentialEvolutionStep<<<POPULATION_SIZE, THREADS_PER_BLOCK>>>(dev_x1, dev_x2, dev_curandState);
        //dev_x2 is the differential model
#if  defined(OPTIMIZER_MIN_INIT_DE) || defined(OPTIMIZER_SIMPLE_DE)
        OPTIMIZER::evaluateF<<<POPULATION_SIZE, THREADS_PER_BLOCK>>>(dev_x2, dev_data, dev_F2, dev_globalContext);
#elif defined(OPTIMIZER_MIN_DE)
        OPTIMIZER::optimize<<<POPULATION_SIZE, THREADS_PER_BLOCK>>>(dev_x2, dev_data, dev_F2, dev_globalContext);
#elif
        std::cerr<<"Incorrect optimizer configuration"<<std::endl;
        exit(1);
#endif
        //evaluated differential model into F2
        selectBestModels<<<POPULATION_SIZE, THREADS_PER_BLOCK>>>(dev_x1, dev_x2, dev_F1, dev_F2, i);
        //select the best models from current and differential models
        std::swap(dev_x1, dev_x2);
        std::swap(dev_F1, dev_F2);
        // dev_x1 contains the next models, dev_F1 contains the associated costs
    }
#if defined(OPTIMIZER_SIMPLE_DE) || defined(OPTIMIZER_MIN_INIT_DE)
        printf("\nthreads:%d\n", THREADS_PER_BLOCK);
        printf("\niterations:%d\n", DE_ITERATION_COUNT);
        printf("\nfevaluations: %d\n", DE_ITERATION_COUNT);
#endif
    printBestF<<<1,1>>>(dev_F1,POPULATION_SIZE);

    hipMemcpy(&finalFs, dev_F1, POPULATION_SIZE * sizeof(double), hipMemcpyDeviceToHost);
    int min=0;
    for(int ff=1;ff<POPULATION_SIZE;ff++){
        if(finalFs[min]>finalFs[ff]){
            min=ff;
        }
    }
    hipMemcpy(&solution, dev_x1, xSize * sizeof(double), hipMemcpyDeviceToHost);
    printf("\nsolf: %f and solution: ",finalFs[min]);
    for(int ff=X_DIM*min;ff<X_DIM*(min+1)-1;ff++) {
        printf("%f,",solution[ff]);
    }
    printf("%f\n",solution[X_DIM*(min+1)-1]);
    persistBestSNLPModel(&solution[X_DIM*min],X_DIM, std::string("finalModel")+std::string(OPTIMIZER::name)+std::string(".csv"));

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float memcpyMilli = 0;
    hipEventElapsedTime(&memcpyMilli, startCopy, stopCopy);
    float kernelMilli = 0;
    hipEventElapsedTime(&kernelMilli, start, stop);
//    printf("Memcpy,kernel elapsed time (ms): %f,%f\n", memcpyMilli, kernelMilli);
    printf("\ntime ms : %f\n", kernelMilli);


    hipFree(dev_x);
    hipFree(dev_xDE);
    hipFree(dev_data);
    hipFree(dev_F);
    hipFree(dev_FDE);

#ifdef GLOBAL_SHARED_MEM
    hipFree(dev_globalContext);
#endif
}

int main() {
//    testDFloat();
//    testDFuncBFS();
//    testF1();

    testPlaneFitting();
//    testQueue();
//    testDot();
    return 0;
}

// Create the Function concept: ([DDouble a])-> compute parameter index order once (BFS), and propagate derivatives that way
// will have: orderArray[operatorTreeSize] - container indices of parameter in order
//            parameters[maxIndex]-contains references of DDouble parameters
// calculate local stack size limit,
// keep a min heap of size operatorTreeSize and a statistical vector to check for duplicates.
// OPT: keep the orderArray in shared memory, to reduce