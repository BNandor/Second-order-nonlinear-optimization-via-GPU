#include "hip/hip_runtime.h"
#include <iostream>

#define SAFE

#include "core/common/Tests.cuh"
#include <random>

//void testDFloat() {
//    DDouble *dev_c;
//    DDouble *c = (DDouble *) malloc(sizeof(DDouble));
//    unsigned *dev_global_id;
//    hipMalloc((void **) &dev_global_id, sizeof(unsigned));
//    hipMalloc((void **) &dev_c, sizeof(DDouble));
//
//
//    unsigned global_id_val = 0;
//    hipMemcpy(dev_global_id, &global_id_val, sizeof(unsigned), hipMemcpyHostToDevice);
//    testDFloatKernel<<<1, 1>>>(dev_c, dev_global_id);
//
//    hipMemcpy(c, dev_c, sizeof(DDouble), hipMemcpyDeviceToHost);
//    assert(c->value == 36);
//    hipFree(dev_c);
//    hipFree(dev_global_id);
//    free(c);
//}
//
//void testDFuncBFS() {
//
//
//    unsigned *dev_global_id;
//    hipMalloc((void **) &dev_global_id, sizeof(unsigned));
//    unsigned global_id_val = 0;
//    hipMemcpy(dev_global_id, &global_id_val, sizeof(unsigned), hipMemcpyHostToDevice);
//    functionTestsKernel<<<1, 1>>>(dev_global_id);
//    hipFree(dev_global_id);
//
//}
//
//void testF1() {
//    unsigned xSize = 2;
//    double *dev_x;
//    hipMalloc((void **) &dev_x, xSize * sizeof(double));
//    double x[2] = {100.0, 2.0};
//    hipMemcpy(dev_x, &x, xSize * sizeof(double), hipMemcpyHostToDevice);
//    testF1DFloat<<<1, 1>>>(dev_x, xSize);
//    hipFree(dev_x);
//}

void generatePlanePoints(double A, double B, double C, double *data, unsigned pointCount) {
    std::uniform_real_distribution<double> unif(0, 1);
    std::default_random_engine re;
    std::normal_distribution<double> normal(0.0, 1);

    for (int i = 0; i < pointCount; i++) {
        data[i * OBSERVARVATION_DIM] = unif(re);
        data[i * OBSERVARVATION_DIM + 1] = unif(re);
        data[i * OBSERVARVATION_DIM + 2] =
                A * data[i * OBSERVARVATION_DIM] + B * data[i * OBSERVARVATION_DIM + 1] + C + normal(re);
    }
}

void testPlaneFitting() {
    const unsigned xSize = X_DIM;
    const unsigned dataSize = OBSERVARVATION_DIM * OBSERVARVATION_COUNT;
    double *dev_x;
    hipMalloc((void **) &dev_x, xSize * sizeof(double));
    double x[xSize] = {1, 1, 1.0};
    double A = 5.5;
    double B = 99;
    double C = -1;
    double data[dataSize] = {};
    generatePlanePoints(A, B, C, data, OBSERVARVATION_COUNT);
    hipMemcpy(dev_x, &x, xSize * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(dev_const_observations), &data, (OBSERVARVATION_COUNT * OBSERVARVATION_DIM) * sizeof(double), 0,
                       hipMemcpyHostToDevice);
    testPlaneFitting<<<1, 128>>>(dev_x);
    hipFree(dev_x);
}

int main() {
//    testDFloat();
//    testDFuncBFS();
//    testF1();
    testPlaneFitting();
    return 0;
}

// Create the Function concept: ([DDouble a])-> compute parameter index order once (BFS), and propagate derivatives that way
// will have: orderArray[operatorTreeSize] - container indices of parameter in order
//            parameters[maxIndex]-contains references of DDouble parameters
// calculate local stack size limit,
// keep a min heap of size operatorTreeSize and a statistical vector to check for duplicates.
// OPT: keep the orderArray in shared memory, to reduce